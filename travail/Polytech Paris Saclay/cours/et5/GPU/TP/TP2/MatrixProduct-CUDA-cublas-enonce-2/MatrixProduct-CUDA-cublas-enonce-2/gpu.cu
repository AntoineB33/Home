#include "hip/hip_runtime.h"
/*********************************************************************************/
/* Matrix product program for a multi-core CPU and for a many-core GPU           */
/* S. Vialle - December 2024                                                     */
/*********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "main.h"
#include "gpu.h"


/*-------------------------------------------------------------------------------*/
/* GPU symbols and global vars                                                   */
/*-------------------------------------------------------------------------------*/
// Symbols used by all kernels
__device__ T_real GPU_A[SIZE][SIZE];
__device__ T_real GPU_B[SIZE][SIZE];
__device__ T_real GPU_C[SIZE][SIZE];

// New Symbol and vars to call Cublas lib.
__device__ T_real GPU_Ctmp[SIZE][SIZE];   // New matrix buffer

T_real *AdrGPU_A = NULL;                  // Adresses of the symbols
T_real *AdrGPU_B = NULL;
T_real *AdrGPU_C = NULL;
T_real *AdrGPU_Ctmp = NULL; 

hipblasHandle_t cublasHandle;              // Handle on the Cublas lib.


/*-------------------------------------------------------------------------------*/
/* Init and finalize the GPU device.                                             */
/*-------------------------------------------------------------------------------*/
void gpuInit(void)
{
  hipInit(0);
  
  // Extract address of GPU matrix "symbols"
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_A,GPU_A),"GPU_A adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_B,GPU_B),"GPU_B adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_C,GPU_C),"GPU_C adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_Ctmp,GPU_Ctmp),"GPU_Ctmp adr extraction");
  
  // Turn CPU arrays A, B and C into "pinned" memory areas
  /**/
  CHECK_CUDA_SUCCESS(hipHostRegister(A,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(B,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(C,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the C CPU array");
  /**/
  
  // Initialize CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasCreate(&cublasHandle), "Init of the CUBLAS lib handle"); 
}


void gpuFinalize(void)
{
  // Turn "pinned" CPU arrays into std array
  /**/
  CHECK_CUDA_SUCCESS(hipHostUnregister(A),
                     "Turning into std memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(B),
                     "Turning into std memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(C),
                     "Turning into std memory the C CPU array");
  /**/

  // Free CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasDestroy(cublasHandle), "Free the CUBLAS lib");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of CPU input data into GPU symbols                                   */
/*-------------------------------------------------------------------------------*/
void gpuSetDataOnGPU(void)
{
 // Set GPU_A symbol
 CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(GPU_A),A,SIZE*SIZE*sizeof(T_real),0,hipMemcpyHostToDevice),
                    "Transfer A-->GPU_A");

 // Set GPU_B symbol
 CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(GPU_B),B,SIZE*SIZE*sizeof(T_real),0,hipMemcpyHostToDevice),
                    "Transfer B-->GPU_B");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of GPU results into CPU array                                        */
/*-------------------------------------------------------------------------------*/
void gpuGetResultOnCPU(void)
{
 // Get GPU_C symbol
 CHECK_CUDA_SUCCESS(hipMemcpyFromSymbol(C,HIP_SYMBOL(GPU_C),SIZE*SIZE*sizeof(T_real),0,hipMemcpyDeviceToHost),
                    "Transfer GPU_C-->C");
}


/*-------------------------------------------------------------------------------*/
/* Transposition kernel using global memory and registers.                       */
/*-------------------------------------------------------------------------------*/
__global__ void TransposeKernel_v0(T_real *MT, T_real *M, int mLig, int nCol)
{
 int lig = threadIdx.y + blockIdx.y*BLOCK_SIZE_XY_KT0;
 int col = threadIdx.x + blockIdx.x*BLOCK_SIZE_XY_KT0;
 
 if (lig < mLig && col < nCol)
   MT[col*mLig + lig] = M[lig*nCol + col];
}


/*-------------------------------------------------------------------------------*/
/* Transposition kernel using global shared memory.                              */
/*-------------------------------------------------------------------------------*/
__global__ void TransposeKernel_v1(T_real *MT, T_real *M, int mLig, int nCol)
{
 // Natural lig-col coordinates of the thread (leading to coalescent reading)
 int firstLibBlock = blockIdx.y*BLOCK_SIZE_XY_KT1;
 int firstColBlock = blockIdx.x*BLOCK_SIZE_XY_KT1;
 int lig = firstLibBlock + threadIdx.y;
 int col = firstColBlock + threadIdx.x;
 
 // Not natural ligT-colT coordinates of the threadin the transpose matrix
 // in order to achieve coalescent writting
 int ligT = firstColBlock + threadIdx.y;
 int colT = firstLibBlock + threadIdx.x;
 
 // shared memory matrix block
 __shared__ T_real shM[BLOCK_SIZE_XY_KT1][BLOCK_SIZE_XY_KT1];
 
 // Coalescent reading of M matrix, and storage in shm matrix block
 if (lig < mLig && col < nCol)
   shM[threadIdx.y][threadIdx.x] = M[lig*nCol + col];
 // Waiting all data are stored in the shm block matrix
 __syncthreads();
 // Coalescent writing in the transposed matrix
 if (ligT < nCol && colT < mLig)
   MT[(firstColBlock + threadIdx.y)*mLig + 
      (firstLibBlock + threadIdx.x)        ] = shM[threadIdx.x][threadIdx.y];
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 1D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v0(void)
{
 // Index computations
 int lig = blockIdx.y;
 int col = threadIdx.x + blockIdx.x*BLOCK_SIZE_X_K0;

 // Matrix product computation
 if (col < SIZE /* && lig < SIZE */) {
   T_real accu = 0.0;
   for (int k = 0; k < SIZE; k++) {
     accu += GPU_A[lig][k]*GPU_B[k][col];
   }
   GPU_C[lig][col] = accu;
 }
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 2D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v1(void)
{
 // Index computations
 int lig = threadIdx.y + blockIdx.y*BLOCK_SIZE_Y_K1;
 int col = threadIdx.x + blockIdx.x*BLOCK_SIZE_X_K1;

 // Matrix product computation
 if (col < SIZE  && lig < SIZE) {
   T_real accu = 0.0;
   for (int k = 0; k < SIZE; k++) {
     accu += GPU_A[lig][k]*GPU_B[k][col];
   }
   GPU_C[lig][col] = accu;
 }
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU.                                        */
/*-------------------------------------------------------------------------------*/
void gpuProduct(gkid_t kid)
{
 dim3 Dg = {0,0,0};   // Grid descriptor
 dim3 Db = {0,0,0};   // Block descriptor
 
 //T_real alpha;        // When using CUBLAS
 //T_real beta;         // When using CUBLAS

 switch(kid) {

  case GK0 : // Kernel v0 - 1D kernel using only resgisters and cache with generic matrix size
   // - init the grid of blocs
   Db.x = BLOCK_SIZE_X_K0;
   Db.y = 1;
   Db.z = 1;
   Dg.x = SIZE/BLOCK_SIZE_X_K0 + (SIZE%BLOCK_SIZE_X_K0 > 0 ? 1 : 0);
   Dg.y = SIZE;
   Dg.z = 1;
   // - run the Grid of Blocs of threads
   MatrixProductKernel_v0<<<Dg,Db>>>();
   break;

 case GK1 : // kernel v1 : 2D kernel using only registers and cache with generic matrix size
   // - init the grid of blocs
   Db.x = BLOCK_SIZE_X_K1;
   Db.y = BLOCK_SIZE_Y_K1;
   Db.z = 1;
   Dg.x = SIZE/BLOCK_SIZE_X_K1 + (SIZE%BLOCK_SIZE_X_K1 > 0 ? 1 : 0);
   Dg.y = SIZE/BLOCK_SIZE_Y_K1 + (SIZE%BLOCK_SIZE_Y_K1 > 0 ? 1 : 0);;
   Dg.z = 1;
   // - run the Grid of Blocs of threads
   MatrixProductKernel_v1<<<Dg,Db>>>();
   break;

 case GK2 : // kernel v2 : 2D kernel using the shared memories
   break;
  
 case GK3 : // kernel v3 : 2D kernel using the shared memories with generic matrix size
   break;

 case GK4 : // calling cublas gemm & user-defined transpose kernel
   break;
   
 case GK5 : // Calling cublas gemm & cublas geam kernels
   break;

 case GK6 : // Calling cublas gemm, using matrix math properties
   break;

 case GK7 : // Calling cublas gemmEx with std 32F/64F datatypes, using Tensor cores
   break;

 case GK8 : // Calling cublas gemmEx with different low precisions on 32/16 bits datatypes, 
            // using Tensor cores
   break;

 default :
   fprintf(stderr,"Unknown GPU kernel!");
   exit(EXIT_FAILURE);
 } // End of switch
}




