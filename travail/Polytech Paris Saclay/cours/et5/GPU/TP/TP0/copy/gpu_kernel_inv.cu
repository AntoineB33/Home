#include "hip/hip_runtime.h"
#include "gpu_kernel.h"


// ----------------------------------------------------------------------------
// GPU blur kernel
// ----------------------------------------------------------------------------
__global__ void gpuBlur(
        unsigned char* input_image, 
        unsigned char* output_image, 
        unsigned int width, 
        unsigned int height) {

    // The offset defines the pixel that will be computed by one thread
    // const unsigned int offset = blockIdx.y*blockDim.x + threadIdx.x;
    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;

    // x and y represent the position of the pixel in the image
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = FSIZE; // Size of the filter to use for bluring

    // Making sure we are only computing pixels inside the image
    if(offset < width*height) {
        // Temporary variables to accumulate the values of RGB in the neighborhood
        unsigned int output_red = 0;
        unsigned int output_green = 0;
        unsigned int output_blue = 0;
        unsigned int hits = 0; // Number of valid pixels (insize the image) used

        // Iterates over the pixels in the neighborhood
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if(x + ox >= 0 && x + ox < width && y + oy >= 0 && y + oy < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += 255 - input_image[currentoffset]; 
                    output_green += 255 - input_image[currentoffset+1];
                    output_blue += 255 - input_image[currentoffset+2];
                    hits++;
                }
            }
        }


        // The output value is the average from the neighborhood
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
    }
}


// ----------------------------------------------------------------------------
// Launching a filtering on GPU
// ----------------------------------------------------------------------------
unsigned char* gpuFilter(
        unsigned char* input_image, 
        unsigned char* output_image, 
        unsigned int width, 
        unsigned int height, 
        unsigned int filter_level) {

    std::cout << "Running the filter on GPU " << filter_level 
        << " times for an image of dimensions " << width << " x " << height << std::endl;

    // References to the data structures in the GPU
    unsigned char* dev_input;
    unsigned char* dev_output;

    // Memory allocation and copy to the GPU
    // For each pixel in width*height, we have 3 colors (RGB)
    getError(hipMalloc((void**) &dev_input, width*height*3*sizeof(unsigned char)),__LINE__);
    getError(hipMemcpy(dev_input, input_image, width*height*3*sizeof(unsigned char),
                hipMemcpyHostToDevice),__LINE__);
    // Memory allocation of the output image in the GPU (no need to copy data)
    getError(hipMalloc((void**) &dev_output, width*height*3*sizeof(unsigned char)),__LINE__);


    dim3 blockDims(BLOCK_SIZE_1,1,1);
    dim3 gridDims((unsigned int) ceil(width*height*3/((double)BLOCK_SIZE_2)), 1, 1 );

    for(unsigned int i = 0; i < filter_level; ++i) {
        //Function call
        gpuBlur<<<gridDims, blockDims>>>(dev_input, dev_output, width, height); 

        //Swaps references except on the last iteration
        if (i < filter_level-1){
            // reference swap
            unsigned char* swap_tmp = dev_output;
            dev_output = dev_input;
            dev_input = swap_tmp;
        }
    }

    // Copies data back from the GPU
    getError(hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char),
                hipMemcpyDeviceToHost),__LINE__);
    // Frees allocated memory in the GPU
    getError(hipFree(dev_input),__LINE__);
    getError(hipFree(dev_output),__LINE__);

    return output_image;
}


// ----------------------------------------------------------------------------
// Error reporting 
// ----------------------------------------------------------------------------
void getError(hipError_t err, int line) {
    if(err != hipSuccess) {
        std::cout << "!! Error on a CUDA function call on line " 
            << line << " : " << hipGetErrorString(err) << std::endl;
    }
}

